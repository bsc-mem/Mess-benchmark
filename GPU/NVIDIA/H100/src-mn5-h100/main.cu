/*
 * Copyright (c) 2024, Barcelona Supercomputing Center
 * Contact: pouya.esmaili    [at] bsc [dot] es
 *          petar.radojkovic [at] bsc [dot] es
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright notice,
 *       this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 *     * Neither the name of the copyright holder nor the names
 *       of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <iostream>
#include <getopt.h>
#include <stdio.h>
#include <unistd.h>
#include <papi.h>

#include "gpuKernels.cuh"
#include "cpuPreprocessing.h"

using namespace std;
#include <chrono>

#define NUM_EVENTS 4

void handle_error(char *where, int rc)
{
    char *str = PAPI_strerror(rc);
    fprintf(stderr, "%s error %d: %s\n", where, rc, str);
    exit(1);
}

uint64_t device_sync = 0;

int main(int argc, char **argv)
{

    /*          PAPI section           */
    // TODO add ifdef PAPI to generalize the code.
    const char *EventName[] = {"cuda:::fbpa__dram_read_bytes.sum.per_second:device=0", "cuda:::fbpa__dram_write_bytes.sum.per_second:device=0", "cuda:::fbpa__dram_read_bytes.sum:device=0", "cuda:::fbpa__dram_write_bytes.sum:device=0"};

    /* Initialize the PAPI library */
    int events[NUM_EVENTS];
    long long values[NUM_EVENTS];
    int retval;
    int EventSet = PAPI_NULL;
    
    validate(PAPI_library_init(PAPI_VER_CURRENT), PAPI_VER_CURRENT);

    /* convert PAPI native events to PAPI code */
    for (uint64_t i = 0; i < NUM_EVENTS; i++) 
    {
        validate(PAPI_event_name_to_code(EventName[i], &events[i]), PAPI_OK);
    }

    validate(PAPI_create_eventset(&EventSet), PAPI_OK);
    validate(PAPI_add_events(EventSet, events, NUM_EVENTS), PAPI_OK);

    const char *filename;            // ptr chase file address
    uint64_t array_bytes_ptrChase;   // size of the ptr chase array in bytes
    uint64_t number_of_instructions; // number of instructions to be executed by pointer chase kernel

    // create two array for our workload generator
    STREAM_TYPE *stream_a; // for reads
    STREAM_TYPE *stream_b; // for writes
    // create two array for our workload generator (GPU side)
    STREAM_TYPE *d_stream_a;  // for reads
    STREAM_TYPE *d_stream_b;  // for writes
    uint64_t rd_percentage, pause; // workload generator inputs to determine r/w ratio and intensity of the workload

    // parse command line arguments and initialize variables
    parseCommandLineArguments(argc, argv, array_bytes_ptrChase, filename, rd_percentage, pause, number_of_instructions);


    cout << "array_bytes_ptrChase: " << array_bytes_ptrChase << endl;
    // create a pointer chase array (each elelemnt is one cacheline)
    struct line *ptrChaceArray = createPointerChaseArray(array_bytes_ptrChase, filename);
    // testPtrChaseCPUPower(ptrChaceArray, number_of_instructions);
    
    struct line *d_ptrChaceArray;

    // copy the pointer chase array for the GPU
    validate(hipMalloc((void **)&d_ptrChaceArray, array_bytes_ptrChase), 0);

    // create workload generator arrays for read (stream_a) and write (stream_b)
    uint64_t workloadGeneratorArraySize = STREAM_ARRAY_SIZE * sizeof(STREAM_TYPE);
    validate(hipHostMalloc((void **)&stream_a, workloadGeneratorArraySize), 0);
    validate(hipHostMalloc((void **)&stream_b, workloadGeneratorArraySize), 0);

    createWorkloadGeneratorArray(stream_a, stream_b);

    // copy the workload generator arrays for the GPU
    validate(hipMalloc((void **)&d_stream_a, workloadGeneratorArraySize), 0);
    validate(hipMalloc((void **)&d_stream_b, workloadGeneratorArraySize), 0);

    // set the kernel function for the workload generator based on the read/write ratio
    setKernelFunction(rd_percentage);
    
    // print GPUs info
    // printGPUInfo(); 

    // chose GPU number zero for our experiments
    // Later we can exploit multiple GPUs to fisnish all the Mess experimental points faster.
    hipSetDevice(0); 

    // get the range of stream priorities for this device
    int priority_high, priority_low;
    hipDeviceGetStreamPriorityRange(&priority_low, &priority_high);

    // create streams with highest and lowest available priorities
    hipStream_t streamPtrChase, streamWorkloadGenerator;
    hipEvent_t startEvent, stopEvent;

    // create two streams with highest and lowest priorities
    // Create two streams and two events for our two kernels (ptrChase and workload generator)
    hipStreamCreateWithPriority(&streamPtrChase, hipStreamNonBlocking, priority_high);         // stream1: ptrChase
    hipStreamCreateWithPriority(&streamWorkloadGenerator, hipStreamNonBlocking, priority_low); // stream2: workload generator
    // check priority_low -> priority_high + 1 
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // copy the pointer chase array to the GPU
    hipMemcpyAsync(d_ptrChaceArray, ptrChaceArray, array_bytes_ptrChase, hipMemcpyHostToDevice, streamPtrChase);
    // copy the workload generator arrays to the GPU
    hipMemcpyAsync(d_stream_a, stream_a, workloadGeneratorArraySize, hipMemcpyHostToDevice, streamWorkloadGenerator);
    hipMemcpyAsync(d_stream_b, stream_b, workloadGeneratorArraySize, hipMemcpyHostToDevice, streamWorkloadGenerator);

    // wait for the data movement to finish
    hipStreamSynchronize(streamPtrChase); 
    hipStreamSynchronize(streamWorkloadGenerator); 

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // get the property of the first GPU
    uint64_t NumberOfBlocksWG = deviceProp.multiProcessorCount-1;
    
    // reset papi counters
    PAPI_reset(EventSet);
    // start the PAPI counters (because of some reason in MN5 we cannot instrument with PAPI while some curves are running)
    retval = PAPI_start(EventSet); 
    if (retval != PAPI_OK)
        fprintf(stderr, "PAPI_start failed\n");

    auto start_bw = std::chrono::system_clock::now();
    // launch the kernels
    // 1. workload generator kernel
    STREAM_copy_rw<<<NumberOfBlocksWG, 1024, 0, streamWorkloadGenerator>>>(d_stream_a, d_stream_b, workloadGeneratorArraySize, pause);
    validate(hipGetLastError(), hipSuccess);
    sleep(20);
    

    // start the timer
    // hipEventRecord(startEvent, 0);
    // reset papi counters
    PAPI_reset(EventSet);
    // start the PAPI counters
    // retval = PAPI_start(EventSet); 
    // if (retval != PAPI_OK)
    //     fprintf(stderr, "PAPI_start failed\n");

    auto start = std::chrono::system_clock::now();


    // 2. ptrChase kernel (1 thread and 1 block)
    pointerChase<<<1, 1, 0, streamPtrChase>>>(d_ptrChaceArray, array_bytes_ptrChase, number_of_instructions);
    validate(hipGetLastError(), hipSuccess);

    // wait for the kernels to finish
    hipStreamSynchronize(streamPtrChase);
    // if(device_sync)
    //     hipStreamSynchronize(streamWorkloadGenerator);

    

    auto end = std::chrono::system_clock::now();
    auto elapsed = end - start;

    // this is more accurate
    std::cout << "latency time elapsed: " << elapsed.count() << " ns" << '\n';
    cout << "average access latency: " << elapsed.count() / number_of_instructions << " ns" << endl;

    elapsed = end - start_bw;
    std::cout << "BW time elapsed: " << elapsed.count() << " ns" << '\n';
    // this way of timing is not accurate for our case. 
    // float elapsedTime;
    // hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);

    // print the time
    // cout << "Time: " << elapsedTime << " ms" << endl;
    // cout << "average access latency " << 1000000 * (elapsedTime / number_of_instructions) << " ns" << endl; 

    // check PAPI stop and read the counters 
    retval = PAPI_stop(EventSet, NULL);
    if (retval != PAPI_OK)
        fprintf(stderr, "PAPI_read failed\n");

    retval = PAPI_read(EventSet, values);
    if (retval != PAPI_OK)
        fprintf(stderr, "PAPI_read failed\n");

    for (uint64_t i = 0; i < NUM_EVENTS; i++)
        printf("%12lld \t\t --> %s \n", values[i], EventName[i]);

    hipMemcpyAsync(stream_a, d_stream_a, STREAM_ARRAY_SIZE * sizeof(STREAM_TYPE), hipMemcpyDeviceToHost, streamWorkloadGenerator);
    hipMemcpyAsync(stream_b, d_stream_b, STREAM_ARRAY_SIZE * sizeof(STREAM_TYPE), hipMemcpyDeviceToHost, streamWorkloadGenerator);

    hipStreamSynchronize(streamPtrChase);
    hipStreamSynchronize(streamWorkloadGenerator);

    cout << "stream_a[0]: " << stream_a[0] << endl;
    cout << "stream_b[0]: " << stream_b[0] << endl;

    hipStreamDestroy(streamPtrChase);
    hipStreamDestroy(streamWorkloadGenerator);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return 0;
}